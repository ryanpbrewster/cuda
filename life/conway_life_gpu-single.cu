#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "conway_life_gpu-single.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: \"%s\" in %s at %d\n", hipGetErrorString(code), file, line);
        if (abort) {
            exit(code);
        }
    }
}

void freeGameBoard(GameBoard * g) {
    free(g->board);
    free(g->work);
    free(g);
}

GameBoard * newGameBoard(size_t R, size_t C) {
    GameBoard * g = (GameBoard *) malloc(sizeof(GameBoard));
    g->R = R;
    g->C = C;
    size_t const BOARD_BYTES = R*C*sizeof(uint8_t);
    g->board = (uint8_t*) malloc(BOARD_BYTES);
    g->work  = (uint8_t*) malloc(BOARD_BYTES);
    memset(g->board, DEAD, BOARD_BYTES);
    return g;
}

__device__ uint8_t newStatus(uint8_t status, int count) {
    if( status == ALIVE ) {
        return (count < 2 || count > 3)? DEAD : ALIVE;
    } else {
        return (count == 3)? ALIVE : DEAD;
    }
}

char cellCharacter(uint8_t status) {
    return (status == ALIVE)? '#' : '.';
}

__global__ void updateCell(uint8_t * next, uint8_t * cur, size_t R, size_t C) {
    for(int i=blockDim.y*blockIdx.y + threadIdx.y; i < R; i += blockDim.y*gridDim.y) {
        for(int j=blockDim.x*blockIdx.x + threadIdx.x; j < C; j += blockDim.x*gridDim.x) {
            // neighbors of (i,j)
            int ns[] = { C*((i-1+R)%R)  + ((j-1+C)%C)
                       , C*((i-1+R)%R)  + ((j  +C)%C)
                       , C*((i-1+R)%R)  + ((j+1+C)%C)
                       , C*((i  +R)%R)  + ((j-1+C)%C)
                       , C*((i  +R)%R)  + ((j+1+C)%C)
                       , C*((i+1+R)%R)  + ((j-1+C)%C)
                       , C*((i+1+R)%R)  + ((j  +C)%C)
                       , C*((i+1+R)%R)  + ((j+1+C)%C)
                       };
            int idx = C*i + j;
            int count = cur[ns[0]] + cur[ns[1]] + cur[ns[2]] + cur[ns[3]] + cur[ns[4]] + cur[ns[5]] + cur[ns[6]] + cur[ns[7]];
            next[idx] = newStatus(cur[idx], count);
        }
    }
}

void updateBoard(GameBoard * g, uint8_t * d_board, uint8_t * d_work) {
    size_t const bytes = g->R * g->C * sizeof(uint8_t);
    gpuErrchk( hipMemcpy(d_board, g->board, bytes, hipMemcpyHostToDevice) );
    updateCell<<<dim3(16,16,1), dim3(16,16,1)>>>(d_work, d_board, g->R, g->C);
    gpuErrchk( hipMemcpy(g->board, d_work, bytes, hipMemcpyDeviceToHost) );
}

void printBoard(GameBoard * g) {
    for(int i=0; i < g->R; i++) {
        for(int j=0; j < g->C; j++) {
            printf("%c", cellCharacter(g->board[g->C*i+j]));
        }
        printf("\n");
    }
}

int main(int argc, char** argv) {
    if( argc != 2 ) {
        fprintf(stderr, "Usage: %s [# of generations to simulate]\n", argv[0]);
        return 1;
    }
    int t = atoi(argv[1]);

    GameBoard * g = newGameBoard(15, 60);

    // Create the R-pentomino somewhere near the center of the board
    int mi = g->R/2;
    int mj = g->C/2;
    g->board[g->C*(mi+0)+(mj+1)] = ALIVE;
    g->board[g->C*(mi+0)+(mj+2)] = ALIVE;
    g->board[g->C*(mi+1)+(mj+0)] = ALIVE;
    g->board[g->C*(mi+1)+(mj+1)] = ALIVE;
    g->board[g->C*(mi+2)+(mj+1)] = ALIVE;

    uint8_t * d_board;
    uint8_t * d_work;
    size_t const bytes = g->R * g->C * sizeof(uint8_t);
    gpuErrchk( hipMalloc(&d_board, bytes) );
    gpuErrchk( hipMalloc(&d_work,  bytes) );

    for(int i=1; i <= t; i++) {
        updateBoard(g, d_board, d_work);
    }
    printBoard(g);

    gpuErrchk( hipFree(d_board) );
    gpuErrchk( hipFree(d_work) );
    freeGameBoard(g);
    return 0;
}

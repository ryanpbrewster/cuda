// metropolis1.cu
/*
 * A simple CUDA-enabled program that approximates Pi by evaluating
 *     Integrate[ 4*x^2*y^2*Exp[-(x^2+y^2)], {x,-inf,inf}, {y,-inf,inf} ] == Pi
 * using Metropolis Monte Carlo, with the weight function Exp[-(x^2+y^2)]
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__device__ float weightFunction(float x, float y) {
    const float k = 0.2;
    const float norm = M_PI/k;
    return exp(-k*(x*x+y*y)) / norm;
}
__device__ float f(float x, float y) {
    return 4*x*x*y*y*exp(-(x*x+y*y));
}





__global__ void initThreads(float* d_out, hiprandState_t* states, float* xs, float* ys, float* radii) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(idx, idx, 0, &states[idx]);
    d_out[idx] = 0.0;
    xs[idx] = 0.0;
    ys[idx] = 0.0;
    radii[idx] = 1.0;
}


__global__ void pi(float* d_out, hiprandState_t* states, float* xs, float* ys, float* radii, int N_TRIALS) {
    const int ITERS_PER_RADIUS_ADJ = 100;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t tmp_state = states[idx];
    float x = xs[idx];
    float y = ys[idx];
    float r = radii[idx];

    int accept = 0, reject = 0;
    float estimate = 0.0;
    float cur_weight = weightFunction(x,y);
    for(int i=1; i <= N_TRIALS; i++) {
        float xp = x + r*(2*hiprand_uniform(&tmp_state)-1);
        float yp = x + r*(2*hiprand_uniform(&tmp_state)-1);
        float new_weight = weightFunction(xp, yp);
        if( hiprand_uniform(&tmp_state) < new_weight/cur_weight ) {
            x = xp;
            y = yp;
            cur_weight = new_weight;
            accept++;
        } else {
            reject++;
        }
        estimate += f(x,y) / cur_weight;

        if( i%ITERS_PER_RADIUS_ADJ == 0 ) {
            float accept_ratio = float(accept) / float(accept + reject);
            float adj = min(max(2.0*accept_ratio, 0.9), 1.1);
            r *= adj;
            accept = 0; reject = 0;
        }
    }

    states[idx] = tmp_state;
    xs[idx] = x;
    ys[idx] = y;
    radii[idx] = r;
    d_out[idx] += estimate / N_TRIALS;
}

int main(int argc, char** argv) {
    int GRID_SIZE  = 256;
    int BLOCK_SIZE = 256;
    int N_TRIALS   = 1000;
    int N_RUNS     = 10;

    char x;
    opterr = 0;
    while((x = getopt(argc, argv, "g:b:t:r:")) != -1) {
        switch(x) {
            case 'g': GRID_SIZE  = atoi(optarg); break;
            case 'b': BLOCK_SIZE = atoi(optarg); break;
            case 't': N_TRIALS   = atoi(optarg); break;
            case 'r': N_RUNS     = atoi(optarg); break;
            case '?':
                if (optopt == 'g' || optopt == 'b' || optopt == 't' || optopt == 'r') {
                    fprintf (stderr, "Option -%c requires an argument.\n", optopt);
                } else {
                    fprintf (stderr, "Unknown option `-%c'.\n", optopt);
                }
                abort();
            default: abort();
        }
    }

    printf("G=% 4d, B=% 4d, T=% 8d, R=% 5d\n", GRID_SIZE, BLOCK_SIZE, N_TRIALS, N_RUNS);

    int N_KERNELS = GRID_SIZE * BLOCK_SIZE;

    float* h_pis = (float*) malloc(N_KERNELS*sizeof(float));
    float* d_pis;
    gpuErrchk(hipMalloc(&d_pis, N_KERNELS * sizeof(float)));

    hiprandState_t* states;
    gpuErrchk(hipMalloc(&states, N_KERNELS * sizeof(hiprandState_t)));

    float* xs;
    gpuErrchk(hipMalloc(&xs, N_KERNELS * sizeof(float)));

    float* ys;
    gpuErrchk(hipMalloc(&ys, N_KERNELS * sizeof(float)));

    float* radii;
    gpuErrchk(hipMalloc(&radii, N_KERNELS * sizeof(float)));

    time_t start = clock();

    initThreads<<<GRID_SIZE, BLOCK_SIZE>>>(d_pis, states, xs, ys, radii);
    for(int irun=1; irun <= N_RUNS; irun++) {
        pi<<<GRID_SIZE, BLOCK_SIZE>>>(d_pis, states, xs, ys, radii, N_TRIALS);
    }
    gpuErrchk(hipMemcpy(h_pis, d_pis, N_KERNELS*sizeof(float), hipMemcpyDeviceToHost));
    float avg = 0.0;
    for(int i=0; i < N_KERNELS; i++) {
        avg += h_pis[i] / N_RUNS;
    }
    avg /= N_KERNELS;

    time_t end = clock();

    int64_t iters = int64_t(N_KERNELS)*int64_t(N_TRIALS)*int64_t(N_RUNS);
    int elapsed = 1000*(end-start)/CLOCKS_PER_SEC;

    cout << "pi = " << avg << "\n";
    cout << elapsed << " ms" << "\n";
    cout << float(iters)/float(elapsed) << " iters/ms\n";

    free(h_pis);
    hipFree(d_pis);
    hipFree(states);
    hipFree(xs);
    hipFree(ys);
    return 0;
}

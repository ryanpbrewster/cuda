
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

int main() {
    int n = 20;
    float* h_xs;
    float* d_xs;

    h_xs = (float*)malloc(n*sizeof(float));
    hipMalloc(&d_xs, n*sizeof(float));


    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MTGP32); // single-precision
    hiprandSetPseudoRandomGeneratorSeed(prng, 42);
    hiprandGenerateUniform(prng, d_xs, n);
    hiprandDestroyGenerator(prng);

    hipMemcpy(h_xs, d_xs, n*sizeof(float), hipMemcpyDeviceToHost);
    for(int i=0; i < n; i++) {
        printf("%f\n", h_xs[i]);
    }

    hipFree(d_xs);
    free(h_xs);

    return 0;
}

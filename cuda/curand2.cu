// curand2.cu
/*
 * A simple CUDA-enabled program that generates random numbers on-the-fly
 * within each kernel.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void rnd(hiprandState_t* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(0, idx, 0, &states[idx]);

    printf("Thread (%d,%d) --> %f\n", blockIdx.x, threadIdx.x, hiprand_uniform(&states[idx]));
}

int main() {
    const int N_BLOCKS = 1;
    const int N_THREADS = 16;
    const int N_KERNELS = N_BLOCKS * N_THREADS;

    hiprandState_t* states;
    hipMalloc(&states, N_KERNELS * sizeof(hiprandState_t));

    rnd<<<N_BLOCKS, N_THREADS>>>(states);

    hipFree(states);
    return 0;
}
